#include <hip/hip_runtime.h>

#include <stdio.h>
#include <time.h>

#define SIZE 10

__global__ void min(int *input)
{
	int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;
	while(number_of_threads>0){
		if(tid<number_of_threads){
			int first = tid*step_size*2;
			int second = first+step_size;
			if(input[second]<input[first])
				input[first]=input[second];
		}
		step_size*= 2;
		number_of_threads/=2;
	}
}

__global__ void max(int *input)
{
	int tid = threadIdx.x;
        int step_size = 1;
        int number_of_threads = blockDim.x;
        while(number_of_threads>0){ 
                if(tid<number_of_threads){ 
                        int first = tid*step_size*2;
                        int second = first+step_size;
                        if(input[second]>input[first])
                                input[first]=input[second];
                }
                step_size*= 2;
                number_of_threads/=2;
        }
}

__global__ void summation(int *input)
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;
	while(number_of_threads>0){
		if(tid<number_of_threads){
			const int first = tid*step_size*2;
			const int second = first+step_size;
			input[first] +=	input[second];
		}
		step_size*=2;
		number_of_threads/=2;
	}
}

__global__ void average(int *input)
{
	const int tid = threadIdx.x;
        int step_size = 1;
        int number_of_threads = blockDim.x;
        while(number_of_threads>0){
                if(tid<number_of_threads){
                        const int first = tid*step_size*2;
                        const int second = first+step_size;
                        input[first] += input[second];
                }
                step_size*=2;
                number_of_threads/=2;
        }
	input[0] = input[0]/10;
}

__global__ void standardDeviation(int *input,int mean)
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;
	int std = 0;
	while(number_of_threads>0){
		if(tid<number_of_threads){
			const int first = tid*step_size*2;
			const int second = first+step_size;
			std = ((input[first]-mean)*(input[first]-mean))+((input[second]-mean)*(input[second]-mean));
		}
		step_size*=2;
		number_of_threads/=2;
	}
	input[0] = std;
}

int main()
{
	int input[SIZE],i;
	for( i = 0 ; i < SIZE ; i++)
	{
		input[i] = rand()% 100;
	}
	for( i = 0 ; i < SIZE ; i++)
	{
		printf("%d ",input[i]);
	}
	printf("\n");
	int byte_size = SIZE*sizeof(int);

	//Allocate mem for min
	//<<<blcoksPerGrid,threadsPerBlock>>>
	int *arr_min, result_min;
	hipMalloc(&arr_min,byte_size);
	hipMemcpy(arr_min,input,byte_size,hipMemcpyHostToDevice);
	min<<<1,SIZE/2>>>(arr_min);
	hipMemcpy(&result_min,arr_min,sizeof(int),hipMemcpyDeviceToHost);
	printf("Minimun: %d\n",result_min);
	
	//Allocate mem for max
	int *arr_max, result_max;
	hipMalloc(&arr_max,byte_size);
	hipMemcpy(arr_max,input,byte_size,hipMemcpyHostToDevice);
	max<<<1,SIZE/2>>>(arr_max);
	hipMemcpy(&result_max,arr_max,sizeof(int),hipMemcpyDeviceToHost);
	printf("Maximum: %d\n",result_max);

	//Allocate mem for sum
	int *arr_sum, sum;
	hipMalloc(&arr_sum,byte_size);
	hipMemcpy(arr_sum,input,byte_size,hipMemcpyHostToDevice);
	summation<<<1,SIZE>>>(arr_sum);
	hipMemcpy(&sum,arr_sum,sizeof(int),hipMemcpyDeviceToHost);
	printf("Sum: %d\n",sum);

	//Allocate mem for avg
	int *arr_avg, avg;
	hipMalloc(&arr_avg,byte_size);
	hipMemcpy(arr_avg,input,byte_size,hipMemcpyHostToDevice);
	//<<<blcoksPerGrid,threadsPerBlock>>>
	average<<<1,SIZE>>>(arr_avg);
	hipMemcpy(&avg,arr_avg,sizeof(int),hipMemcpyDeviceToHost);
	printf("Average: %d\n",avg);
	printf("CPUAVG: %d\n",(sum/SIZE));
	
	//Allcate mem for std
	int *arr_std, std;
	const int mean = avg;
	hipMalloc(&arr_std,byte_size);
	hipMemcpy(arr_std,input,byte_size,hipMemcpyHostToDevice);
	standardDeviation<<<1,SIZE>>>(arr_std,mean);
	hipMemcpy(&std,arr_std,sizeof(int),hipMemcpyDeviceToHost);
	std = sqrt(std/10);
	printf("Standard Deviation: %d\n",std);	

	return 0;
}

